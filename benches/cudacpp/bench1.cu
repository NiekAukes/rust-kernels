#include <iostream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error in " << __FILE__ << " at line " << __LINE__ << " : " << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

__global__ void matrix_mul(const int* a, const int* b, int* c, int n) {
    int t = blockIdx.x * blockDim.x + threadIdx.x;
    int i = t / n;
    int j = t % n;
    
    if (i < n && j < n) {
        int sum = 0;
        for (int k = 0; k < n; ++k) {
            sum += a[i * n + k] * b[k * n + j];
        }
        c[i * n + j] = sum;
    }
}

void matrix_mul_cpu(const std::vector<int>& a, const std::vector<int>& b, std::vector<int>& c, int n) {
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            int sum = 0;
            for (int k = 0; k < n; ++k) {
                sum += a[i * n + k] * b[k * n + j];
            }
            c[i * n + j] = sum;
        }
    }
}

const int ns[] = { 4, 16, 64, 256, 512, 1024, 2048, 4096, 8192 };

int main() {
    for (int s = 0; s < 9; ++s) {
        int n = ns[s];
        size_t size = n * n * sizeof(int);

        std::cout << "Matrix size: " << n << "x" << n << std::endl;

        std::vector<int> a(n * n), b(n * n), c(n * n, 0), c_cpu(n * n, 0);
        
        for (int i = 0; i < n * n; ++i) {
            a[i] = (i * 1234567) % 1000;
            b[i] = (i * 7654321) % 1000;
        }

        auto gpu_start = std::chrono::high_resolution_clock::now();
        int *d_a, *d_b, *d_c;
        CHECK_CUDA(hipMalloc(&d_a, size));
        CHECK_CUDA(hipMalloc(&d_b, size));
        CHECK_CUDA(hipMalloc(&d_c, size));

        CHECK_CUDA(hipMemcpy(d_a, a.data(), size, hipMemcpyHostToDevice));
        CHECK_CUDA(hipMemcpy(d_b, b.data(), size, hipMemcpyHostToDevice));

        int threads_per_block = 64;
        int blocks = (n * n + threads_per_block - 1) / threads_per_block;

    
        matrix_mul<<<blocks, threads_per_block>>>(d_a, d_b, d_c, n);
        CHECK_CUDA(hipGetLastError());

        CHECK_CUDA(hipMemcpy(c.data(), d_c, size, hipMemcpyDeviceToHost));

        auto gpu_end = std::chrono::high_resolution_clock::now();
        auto gpu_time = std::chrono::duration_cast<std::chrono::milliseconds>(gpu_end - gpu_start).count();

        std::cout << "GPU time: " << gpu_time << " ms" << std::endl;

        CHECK_CUDA(hipFree(d_a));
        CHECK_CUDA(hipFree(d_b));
        CHECK_CUDA(hipFree(d_c));
    }
    return 0;
}
